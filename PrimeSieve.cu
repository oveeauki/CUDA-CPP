#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "chrono"

#define N 99999990

__global__ void sieve(int *primes, int n) {
    int i, j;
    for (i = blockIdx.x * blockDim.x + threadIdx.x + 2; i <= n; i += blockDim.x * gridDim.x) {
        if (primes[i]) {
            for (j = i * i; j <= n; j += i) {
                primes[j] = 0;
            }
        }
    }
}

int main() {
    int *primes, i, count = 0;
    hipMallocManaged(&primes, (N + 1) * sizeof(int));
    for (i = 0; i <= N; i++) {
        primes[i] = 1;
    }
    primes[0] = 0;
    primes[1] = 0;

    int blockSize = 1024;
    int numBlocks = (N + blockSize - 1) / blockSize;
    auto start = std::chrono::high_resolution_clock::now();

    sieve<<<numBlocks, blockSize>>>(primes, N);

    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed_seconds = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();
    auto elapsed_minutes = elapsed_seconds / 60;
    elapsed_seconds = elapsed_seconds % 60;
    for (i = 2; i <= N; i++) {
        if (primes[i]) {
            count++;
        }
    }

    printf("Number of primes up to %d: %d\nElapsed time:%dm %ds\n",N,count,elapsed_minutes,elapsed_seconds);
    hipFree(primes);
    return 0;
}
